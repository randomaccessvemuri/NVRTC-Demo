#include "hip/hip_runtime.h"
﻿//NVRTC DEMO: We take the iteration function for a fractal and compile it into a CUDA kernel at runtime.
const int IMAGE_X = 1920;
const int IMAGE_Y = 1080;
#define randomness 2

#include <hip/hip_runtime.h>
#include <hip/hiprtc.h>
#include <hip/hip_runtime.h>
#include <plog/Log.h>
#include "plog/Initializers/ConsoleInitializer.h"
#include <iostream>
#include <string>
#include <plog/Formatters/TxtFormatter.h>
#include <cuda/std/complex>
#include <fstream>
#include <regex>
#include <imgui.h>
#include <imgui_stdlib.h>
#include <imgui-SFML.h>
#include <SFML/Graphics.hpp>
#include <SFML/Window.hpp>







const std::string ITERATION_FUNCTION = "z = z*z + c";
const char* headerSourceDir[] = {""};
const char* headerNames[] = { "hip/hip_complex.h" };



const std::string code = R"(
#include <cuda/std/complex>
extern "C" __global__ void iterFunc(uchar4 * img, float2 cartesianBounds, float2 cartesianOrigin, int2 imageDims, uchar4* colorMap, int colorMapSize, int maxBounces) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= imageDims.x || y >= imageDims.y) return;

	
	
	float cartesianX = (x - imageDims.x / 2) * cartesianBounds.x / imageDims.x + cartesianOrigin.x;
	float cartesianY = (y - imageDims.y / 2) * cartesianBounds.y / imageDims.y + cartesianOrigin.y;

	cuda::std::complex<float> c(cartesianX, cartesianY);
	cuda::std::complex<float> z = c;
	int bounces = 0;
	
	for (int i = 0; i < maxBounces; i++) {
		ITERATION_FUNCTION;
		if ((z.real() * z.real() + z.imag() * z.imag()) > cartesianBounds.x * cartesianBounds.y) {
			break;
		}
		bounces++;
	}

	if (bounces == maxBounces) {
		img[y * imageDims.x + x] = colorMap[0];
		return;
	}

	


	float bouncesForBlending = (float)bounces + 1 - log(z.real() * z.real() + z.imag() * z.imag())/log(2.f);
	
	float t = abs(bouncesForBlending - (int)bouncesForBlending); // fractional part
	
	int index1 = ((int)bouncesForBlending % (colorMapSize - 1)) + 1;
	int index2 = (index1 % (colorMapSize - 1)) + 1;

	uchar4 color1 = colorMap[index1==0?1: index1];
	uchar4 color2 = colorMap[index2==0?1: index2];

	// Linear interpolation
	uchar4 color;
	color.x = color1.x * (t) + color2.x * (1.0-t);
	color.y = color1.y * (t) + color2.y * (1.0-t);
	color.z = color1.z * (t) + color2.z * (1.0-t);
	color.w = 255;

	img[y * imageDims.x + x] = color;
}
)";

struct imgDims{
	int width;
	int height;
};


struct ptxExecEnv {
	hipDevice_t device;
	hipCtx_t context;
	hipModule_t module;
	hipFunction_t kernel;
	void* args;
	const char* ptx;
	unsigned long long ptxSize;
	int gridSize;
	int blockSize;
	hipDeviceptr_t imgDevice;
	uchar4* imgHost;
	unsigned long long bufferSize;
	hiprtcProgram prog;
};
struct preRenderConfig {
	float2 cartesianBounds;
	float2 cartesianOrigin;
	imgDims imageDims;
	int maxBounces;
	uchar4* colorMap;
	int colorMapSize;
	std::string iterationFunction;
};

ptxExecEnv initCuda() {
	hipInit(0);
	hipDevice_t device;
	hipCtx_t context;

	hipDeviceGet(&device, 0);
	hipCtxCreate(&context, 0, device);

	return {
		device,
		context,
		nullptr,
		nullptr,
		nullptr,
		nullptr,
		0,
		0,
		0,
		0,
		nullptr,
		0,
		0
	};

}

void generatePTX(std::string kernelCode, preRenderConfig preRenderConfigObj, ptxExecEnv* env) {

	kernelCode= std::regex_replace(kernelCode, std::regex("ITERATION_FUNCTION"), preRenderConfigObj.iterationFunction);

	std::cout << "Kernel Code: \n" << kernelCode << "\n";
	_sleep(1000);


	PLOGD << "Generating PTX Code From Kernel Code..";
	hiprtcProgram prog;
	hiprtcCreateProgram(
		&prog, 
		kernelCode.c_str(),
		"iterFunc",
		1,
		headerSourceDir,
		headerNames
	);

	//One of the bugs encountered was here. GTX 1650 doesn't support Compute Capability 8.0 so we need to set it to 7.5
	//TODO: Get the compute capability of the device and set it accordingly
	const char* opts[] = { "--gpu-architecture=compute_52",
		 "--fmad=false", "--include-path=D:/include/"};
	PLOGD << "Compiling Kernel To CUDA PTX..\n";
	if (hiprtcCompileProgram(prog, 3, opts) != HIPRTC_SUCCESS) {
		PLOGE << "Failed to compile NVRTC Program!";
	}

	size_t logSize;
	if (hiprtcGetProgramLogSize(prog, &logSize) != HIPRTC_SUCCESS) {
		PLOGE << "Failed to get NVRTC Log Size!";
	}
	char* log = new char[logSize];
	hiprtcGetProgramLog(prog, log);
	// Obtain PTX from the program.
	size_t ptxSize;
	hiprtcGetCodeSize(prog, &ptxSize);
	std::string ptxLog(log, logSize);
	PLOGW << "Generated NVRTC Log: \n" << ptxLog;
	char* ptx = new char[ptxSize];
	hiprtcGetCode(prog, ptx);
	PLOGD << "PTX Code generated!";
	
	hipModule_t module;
	hipFunction_t kernel;
	hipInit(0);
	if (hipDeviceGet(&env->device, 0)!= hipSuccess) {
		PLOGE << "Failed to get CUDA device!";
	}
	hipCtxCreate(&env->context, 0, env->device);
	auto moduleLoadResult = hipModuleLoadDataEx(&module, ptx, 0, 0, 0);
	PLOGW << "Module Load Result: " << hipGetErrorString(static_cast<hipError_t>(moduleLoadResult));

	auto kernelLoadResult = hipModuleGetFunction(&kernel, module, "iterFunc");
	PLOGW << "Kernel Load Result: " << hipGetErrorString(static_cast<hipError_t>(kernelLoadResult));
	//Kernel load was failing because I didn't use extern C which resulted in name mangling and the kernel not being found post compilation
	unsigned long long n = static_cast<unsigned long long>(preRenderConfigObj.imageDims.width) * static_cast<unsigned long long>(preRenderConfigObj.imageDims.height);
	unsigned long long bufferSize = n * sizeof(uchar4);
	int blockSize = 32;
	int gridSize = (n + blockSize - 1) / blockSize;
	uchar4* hostPtr = new uchar4[bufferSize];
	hipDeviceptr_t imgDevice;
	hipMalloc(&imgDevice, bufferSize);
	hipDeviceptr_t colorMapDevice;
	hipMalloc(&colorMapDevice, preRenderConfigObj.colorMapSize * sizeof(uchar4));
	hipMemcpyHtoD(colorMapDevice, preRenderConfigObj.colorMap, preRenderConfigObj.colorMapSize * sizeof(uchar4));

	void* args[] = {&imgDevice,  &preRenderConfigObj.cartesianBounds, &preRenderConfigObj.cartesianOrigin, &preRenderConfigObj.imageDims, &colorMapDevice, &preRenderConfigObj.colorMapSize, &preRenderConfigObj.maxBounces};

	env->module = module;
	env->kernel = kernel;
	env->ptx = ptx;
	env->ptxSize = ptxSize;
	env->gridSize = gridSize;
	env->blockSize = blockSize;
	env->imgDevice = imgDevice;
	env->imgHost = hostPtr;
	env->bufferSize = bufferSize;
	env->args = args;
	env->prog = prog;
}

void executePTXCode(ptxExecEnv runConfig) {
	//List out everything about the execution environment
	int gridSizeX = (IMAGE_X + runConfig.blockSize - 1) / runConfig.blockSize;
	int gridSizeY = (IMAGE_Y + runConfig.blockSize - 1) / runConfig.blockSize;
	
	auto result = hipModuleLaunchKernel(
		runConfig.kernel,
		gridSizeX, gridSizeY, 1,
		runConfig.blockSize, runConfig.blockSize, 1,
		0,
		NULL,
		(void**)runConfig.args,
		NULL
		);

	PLOGW << "Kernel Launch Result: " << hipGetErrorString(static_cast<hipError_t>(result));
	hipCtxSynchronize();
	PLOGD << "Kernel Executed!";
	hipMemcpyDtoH(runConfig.imgHost, runConfig.imgDevice, runConfig.bufferSize);
}

void terminatePTXEnv(ptxExecEnv runConfig) {
	hipFree(runConfig.imgDevice);
	hipModuleUnload(runConfig.module);
	hipCtxDestroy(runConfig.context);
	hiprtcDestroyProgram(&runConfig.prog);
	delete[] runConfig.imgHost;
	delete[] runConfig.ptx;
}

//Initialize window:



int main() {
	//Initialize the logger
	plog::ColorConsoleAppender<plog::TxtFormatter> consoleAppender;
	plog::init(plog::verbose, &consoleAppender);


	//Setup the image
	imgDims imgConfig = { IMAGE_X, IMAGE_Y };
	float2 cartesianBounds = { 4, 4};
	float2 cartesianOrigin = { 0, 0 };
	int maxBounces = 1500;
	uchar4 thermalColorMap[] = {
	{0, 0, 0, 255}, // Black
	{0, 0, 255, 255}, // Blue
	{0, 255, 255, 255}, // Cyan
	{0, 255, 0, 255}, // Green
	{255, 255, 0, 255}, // Yellow
	{255, 0, 0, 255}, // Red
	{255, 255, 255, 255}, // White
	};

	preRenderConfig config = {
		cartesianBounds,
		cartesianOrigin, 
		imgConfig, 
		maxBounces, 
		thermalColorMap, 
		sizeof(thermalColorMap) / sizeof(uchar4),
		"z = z*z*z*z + c;"
	};

	PLOGD << "Color Map: " << sizeof(thermalColorMap) / sizeof(uchar4) << "colors";

	ptxExecEnv runConfig = initCuda();
	generatePTX(code, config, &runConfig);
	executePTXCode(runConfig);
	uchar4* img = runConfig.imgHost;
	////Write the image to a file
	//std::ofstream imgFile("D:/temp/img.ppm");
	//imgFile << "P3\n" << imgConfig.width << " " << imgConfig.height << "\n255\n";
	//for (int i = 0; i < imgConfig.width * imgConfig.height; i++) {
	//	imgFile << static_cast<int>(img[i].x) << " " << static_cast<int>(img[i].y) << " " << static_cast<int>(img[i].z) << "\n";
	//}


	sf::RenderWindow window(sf::VideoMode(IMAGE_X, IMAGE_Y), "Fractal Explorer");
	sf::Image sfmlImage;
	sfmlImage.create(imgConfig.width, imgConfig.height, reinterpret_cast<sf::Uint8*>(img));
	sf::Texture texture;
	texture.loadFromImage(sfmlImage);
	sf::Sprite sprite;
	sprite.setTexture(texture);

	ImGui::SFML::Init(window);

	sf::Event event;
	sf::Clock deltaClock;
	while (window.isOpen())
	{
		sf::Event event;
		while (window.pollEvent(event)) {
			ImGui::SFML::ProcessEvent(window, event);

			if (event.type == sf::Event::Closed) {
				window.close();
			}
		}

		ImGui::SFML::Update(window, deltaClock.restart());

		ImGui::Begin("Hello, world!");
		ImGui::SliderFloat2("Cartesian Bounds", &config.cartesianBounds.x, 0.0f, 10.0f);
		ImGui::SliderFloat2("Cartesian Origin", &config.cartesianOrigin.x, -10.0f, 10.0f);
		ImGui::SliderInt("Max Bounces", &config.maxBounces, 0, 10000);
		ImGui::InputText("Iteration Function", &config.iterationFunction);
	if (ImGui::Button("Render")) {
			generatePTX(code, config, &runConfig);
			executePTXCode(runConfig);
			sfmlImage.create(imgConfig.width, imgConfig.height, reinterpret_cast<sf::Uint8*>(runConfig.imgHost));
			texture.loadFromImage(sfmlImage);
			sprite.setTexture(texture);

		}
		ImGui::End();

		window.clear();
		window.draw(sprite);
		ImGui::SFML::Render(window);
		window.display();
	}

	ImGui::SFML::Shutdown();

	terminatePTXEnv(runConfig);
	return 0;
}